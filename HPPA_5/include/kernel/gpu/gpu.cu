#include "hip/hip_runtime.h"
#pragma once
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <entity/image.h>
#include "gpu.h"

#define WARP_SIZE 32
#define DEVICE_ID 0

__constant__ unsigned __int32 LAST_BLOCK_HEIGHT;
__constant__ unsigned __int32 FULL_BLOCK_HEIGHT;
__constant__ unsigned __int32 BLOCK_LENGHT_IN_INT;
__constant__ unsigned __int32 BLOCK_LENGHT;
__constant__ unsigned __int32 ITERATIONS_PER_THREAD;
__constant__ unsigned __int32 SRC_PITCH_IN_INT;
__constant__ unsigned __int32 SRC_HEIGHT;
__constant__ unsigned __int32 SRC_WIDTH;
__constant__ unsigned __int32 LAST_PIXEL_INDEX;
__constant__ unsigned __int32 C_PIVOT_ITERATIONS_PER_THREAD;


inline __global__ void _debugKernel(const unsigned __int32* src, unsigned __int32* dst);
inline __global__ void c_debugKernel(const unsigned __int32* src, unsigned __int32* dst);

inline __device__ void sort(const unsigned int** ls_buffer_ptr, unsigned int** ls_sorted_buffer_ptr);
inline __device__ void init_lower_pivot(unsigned char* line, unsigned  char  pivot_window[4]);
inline __device__ void init_median_pivot(unsigned char* line, unsigned  char  pivot_window[4]);
inline __device__ void init_high_pivot(unsigned char* line, unsigned  char  pivot_window[4]);
inline __device__ void c_init_lower_pivot(unsigned char* line, unsigned  char  pivot_window[4]);
inline __device__ void c_init_median_pivot(unsigned char* line, unsigned  char  pivot_window[4]);
inline __device__ void c_init_high_pivot(unsigned char* line, unsigned  char  pivot_window[4]);
inline __device__ void c_pivot_median(unsigned __int32* ls_result_buffer, unsigned __int32* ls_sorted_buffer_ptr[3]);
inline __device__ void pivot_median(unsigned __int32* ls_result_buffer, unsigned __int32* ls_sorted_buffer_ptr[3]);
inline __device__ void store(const unsigned __int32 counter, unsigned int* dst, unsigned int* ls_result_buffer, const unsigned __int32, const unsigned __int32);
inline __device__ void load_line(unsigned int* ls_buffer_ptr, const unsigned int* src, const signed int iter, const unsigned __int32, const unsigned __int32);
inline __device__ void c_load_line(unsigned int* ls_buffer_ptr, const unsigned int* src, const signed int iter, const unsigned __int32, const unsigned __int32);

void cudaHandler_opt(
		const image& src,
		image& dst,
		double& time
		)
{

	const unsigned __int32 full_block_height = 33;	//low dependense to excec time (less then ~6 cause unhandled misses)
	const unsigned __int32 block_length = 2048;
	const unsigned __int32 block_length_in_int = block_length/4;
	dst.magic = src.magic;
	dst.depth = src.depth;
	dst.height = src.height;
	dst.width = src.width;
	dst.data = new char[dst.height * dst.width];
	const unsigned __int32  warps_per_block = 8;
	const unsigned __int32  threads_per_block = WARP_SIZE * warps_per_block;
	const unsigned __int32 blocks_per_raw = (src.width + block_length - 1) / block_length;
	const unsigned __int32 blocks_per_column = (src.height + full_block_height - 1) / full_block_height;
	const unsigned __int32 last_block_height = src.height%full_block_height == 0 ? full_block_height : src.height % full_block_height;
	const unsigned __int32 iterations_per_thread = block_length / 4 / threads_per_block;
	const unsigned __int32 shared_alloc_size = 6 * (block_length+128);
	const unsigned int aligne_length = blocks_per_raw * block_length;
	const unsigned __int32 aligne_length_in_int = aligne_length / 4;
	const unsigned __int32 last_pixel_index = src.width%block_length == 0 ? block_length : src.width % block_length;
	unsigned __int8* cuda_src, *cuda_dst;
	hipError_t cudaStatus;
	
	hipEvent_t start, stop;
	float elapsedTime = 0;

	cudaStatus = hipSetDevice(DEVICE_ID);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipEventCreate(&start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipEventCreate(&stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&cuda_dst, aligne_length * dst.height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&cuda_src, aligne_length * src.height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D((void*)cuda_src, aligne_length, (const void*)src.data, src.width, src.width, src.height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	

	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(FULL_BLOCK_HEIGHT), &full_block_height, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);		//year, that's ridiculous! but i'm done...
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(BLOCK_LENGHT_IN_INT), &block_length_in_int, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(BLOCK_LENGHT), &block_length, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(SRC_PITCH_IN_INT), &aligne_length_in_int, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(ITERATIONS_PER_THREAD), &iterations_per_thread, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(LAST_BLOCK_HEIGHT), &last_block_height, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(SRC_HEIGHT), &src.height, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(SRC_WIDTH), &src.width, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(LAST_PIXEL_INDEX), &last_pixel_index, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed!");
		goto Error;
	}

	dim3 blockDim(threads_per_block);
	dim3 gridDim(blocks_per_raw, blocks_per_column);

	cudaStatus = hipEventRecord(start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	_debugKernel << <gridDim, blockDim, shared_alloc_size >> > ((const unsigned int*)cuda_src, (unsigned int*)cuda_dst);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipEventRecord(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipEventElapsedTime(&elapsedTime, start, stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D((void*)dst.data, dst.width, (const void*)cuda_dst, aligne_length,  dst.width, dst.height, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	fprintf(stderr, hipGetErrorString(cudaStatus));
	hipFree(cuda_dst);
	hipFree(cuda_src);

	time = elapsedTime / 1000;
}

void c_cudaHandler_opt(const image& src, image& dst, double& time)
{
	const unsigned __int32 full_block_height = 33;	//low dependense to excec time (less then ~6 cause unhandled misses)
	const unsigned __int32 block_length = 2048;
	const unsigned __int32 block_length_in_int = block_length / 4;
	dst.magic = src.magic;
	dst.depth = src.depth;
	dst.height = src.height;
	dst.width = src.width;
	dst.data = new char[dst.height * dst.width * 3];
	const unsigned __int32 length_in_byte = src.width * 3;
	const unsigned __int32 warps_per_block = 8;
	const unsigned __int32 threads_per_block = WARP_SIZE * warps_per_block;
	const unsigned __int32 blocks_per_raw = (length_in_byte + block_length - 1) / block_length;
	const unsigned __int32 blocks_per_column = (src.height + full_block_height - 1) / full_block_height;
	const unsigned __int32 last_block_height = src.height % full_block_height == 0 ? full_block_height : src.height % full_block_height;
	const unsigned __int32 iterations_per_thread = block_length / 4 / threads_per_block;
	const unsigned __int32 shared_alloc_size = 6 * (block_length + 128);
	const unsigned int aligne_length = blocks_per_raw * block_length;
	const unsigned __int32 aligne_length_in_int = aligne_length / 4;
	const unsigned __int32 last_pixel_index = length_in_byte % block_length == 0 ? block_length : length_in_byte % block_length;
	const unsigned __int32 c_pivot_iterations_per_thread = block_length_in_int / 3 / threads_per_block;
	unsigned __int8* cuda_src, * cuda_dst;
	hipError_t cudaStatus;

	hipEvent_t start, stop;
	float elapsedTime = 0;

	cudaStatus = hipSetDevice(DEVICE_ID);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipEventCreate(&start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipEventCreate(&stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&cuda_dst, aligne_length * dst.height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&cuda_src, aligne_length * src.height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy2D((void*)cuda_src, aligne_length, (const void*)src.data, length_in_byte, length_in_byte, src.height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}



	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(FULL_BLOCK_HEIGHT), &full_block_height, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);		//year, that's ridiculous! but i'm done...
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(BLOCK_LENGHT_IN_INT), &block_length_in_int, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(BLOCK_LENGHT), &block_length, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(SRC_PITCH_IN_INT), &aligne_length_in_int, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(ITERATIONS_PER_THREAD), &iterations_per_thread, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(LAST_BLOCK_HEIGHT), &last_block_height, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(SRC_HEIGHT), &src.height, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(SRC_WIDTH), &length_in_byte, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	cudaStatus | hipMemcpyToSymbol(HIP_SYMBOL(LAST_PIXEL_INDEX), &last_pixel_index, sizeof(unsigned __int32), 0, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed!");
		goto Error;
	}

	dim3 blockDim(threads_per_block);
	dim3 gridDim(blocks_per_raw, blocks_per_column);

	cudaStatus = hipEventRecord(start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	c_debugKernel << <gridDim, blockDim, shared_alloc_size >> > ((const unsigned int*)cuda_src, (unsigned int*)cuda_dst);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipEventRecord(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipEventElapsedTime(&elapsedTime, start, stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy2D((void*)dst.data, length_in_byte, (const void*)cuda_dst, aligne_length, length_in_byte, dst.height, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	fprintf(stderr, hipGetErrorString(cudaStatus));
	hipFree(cuda_dst);
	hipFree(cuda_src);

	time = elapsedTime / 1000;
}



inline __global__ void _debugKernel(const unsigned __int32* src, unsigned __int32* dst) {
	extern __shared__ unsigned __int32 allocated_space[];
	
	unsigned __int32* ls_buffer_ptr[3];
	ls_buffer_ptr[0] = (unsigned __int32*) allocated_space;
	ls_buffer_ptr[1] = (unsigned __int32*) (allocated_space + BLOCK_LENGHT_IN_INT + 32);
	ls_buffer_ptr[2] = (unsigned __int32*) (allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 2);
	unsigned __int32* ls_sorted_buffer_ptr[3];
	ls_sorted_buffer_ptr[0] = (unsigned __int32*) (allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 3);
	ls_sorted_buffer_ptr[1] = (unsigned __int32*) (allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 4);
	ls_sorted_buffer_ptr[2] = (unsigned __int32*) (allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 5);

	const unsigned __int32 BLOCK_HEIGHT = blockIdx.y == gridDim.y - 1 ?  LAST_BLOCK_HEIGHT : FULL_BLOCK_HEIGHT;
	const unsigned __int32 BLOCK_OFFSET_X = blockIdx.x * BLOCK_LENGHT_IN_INT;
	const unsigned __int32 BLOCK_OFFSET_Y = blockIdx.y * FULL_BLOCK_HEIGHT;

	load_line(ls_buffer_ptr[1], src, -1, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);
	load_line(ls_buffer_ptr[2], src, 0, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);

	for (int counter = 0; counter < BLOCK_HEIGHT; counter++) {
		unsigned __int32* ls_result_buffer = ls_buffer_ptr[(counter+1) % 3];
		
		load_line(ls_buffer_ptr[ counter % 3 ], src, counter + 1, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);
		
		sort(ls_buffer_ptr, ls_sorted_buffer_ptr);

		pivot_median(ls_result_buffer, ls_sorted_buffer_ptr); 
	
		store(counter, dst, ls_result_buffer, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);
	}
}


inline __global__ 
void c_debugKernel(const unsigned __int32* src, unsigned __int32* dst) {
	extern __shared__ unsigned __int32 allocated_space[];

	unsigned __int32* ls_buffer_ptr[3];
	ls_buffer_ptr[0] = (unsigned __int32*)allocated_space;
	ls_buffer_ptr[1] = (unsigned __int32*)(allocated_space + BLOCK_LENGHT_IN_INT + 32);
	ls_buffer_ptr[2] = (unsigned __int32*)(allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 2);
	unsigned __int32* ls_sorted_buffer_ptr[3];
	ls_sorted_buffer_ptr[0] = (unsigned __int32*)(allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 3);
	ls_sorted_buffer_ptr[1] = (unsigned __int32*)(allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 4);
	ls_sorted_buffer_ptr[2] = (unsigned __int32*)(allocated_space + (BLOCK_LENGHT_IN_INT + 32) * 5);

	const unsigned __int32 BLOCK_HEIGHT = blockIdx.y == gridDim.y - 1 ? LAST_BLOCK_HEIGHT : FULL_BLOCK_HEIGHT;
	const unsigned __int32 BLOCK_OFFSET_X = blockIdx.x * BLOCK_LENGHT_IN_INT;
	const unsigned __int32 BLOCK_OFFSET_Y = blockIdx.y * FULL_BLOCK_HEIGHT;

	c_load_line(ls_buffer_ptr[1], src, -1, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);
	c_load_line(ls_buffer_ptr[2], src, 0, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);

	for (int counter = 0; counter < BLOCK_HEIGHT; counter++) {
		unsigned __int32* ls_result_buffer = ls_buffer_ptr[(counter + 1) % 3];

		c_load_line(ls_buffer_ptr[counter % 3], src, counter + 1, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);

		sort(ls_buffer_ptr, ls_sorted_buffer_ptr);

		c_pivot_median(ls_result_buffer, ls_sorted_buffer_ptr);

		store(counter, dst, ls_result_buffer, BLOCK_OFFSET_X, BLOCK_OFFSET_Y);
	}
}

inline __device__ 
void pivot_median(unsigned __int32 * ls_result_buffer, unsigned __int32* ls_sorted_buffer_ptr[3]) {
	for (int counter_1 = 0; counter_1 < ITERATIONS_PER_THREAD; counter_1++) {
		const int OFFSET = blockDim.x * counter_1 + threadIdx.x;
		int ls_line[3];
		int result;
		unsigned char pivot_window[3][4];
		unsigned char* line = (unsigned char*)ls_line;
		
		ls_line[0] = ls_sorted_buffer_ptr[2][OFFSET];
		ls_line[1] = ls_sorted_buffer_ptr[2][OFFSET + 1];
		ls_line[2] = ls_sorted_buffer_ptr[2][OFFSET + 2];
		
		init_lower_pivot(line, pivot_window[0]);

		ls_line[0] = ls_sorted_buffer_ptr[1][OFFSET];
		ls_line[1] = ls_sorted_buffer_ptr[1][OFFSET + 1];
		ls_line[2] = ls_sorted_buffer_ptr[1][OFFSET + 2];
		
		init_median_pivot(line, pivot_window[1]);
		
		ls_line[0] = ls_sorted_buffer_ptr[0][OFFSET];
		ls_line[1] = ls_sorted_buffer_ptr[0][OFFSET + 1];
		ls_line[2] = ls_sorted_buffer_ptr[0][OFFSET + 2];
		
		init_high_pivot(line, pivot_window[2]);

		unsigned int sum = __vadd4(*((int*)(pivot_window[0])), __vadd4(*((int*)(pivot_window[1])), *((int*)(pivot_window[2]))));
		unsigned int min = __vminu4(*((int*)(pivot_window[0])), __vminu4(*((int*)(pivot_window[1])), *((int*)(pivot_window[2]))));
		unsigned int max = __vmaxu4(*((int*)(pivot_window[0])), __vmaxu4(*((int*)(pivot_window[1])), *((int*)(pivot_window[2]))));
		result = __vsub4(sum, __vadd4(min, max));
		
		ls_result_buffer[OFFSET] = result;
	}
}

inline __device__
void c_pivot_median(unsigned __int32* ls_result_buffer, unsigned __int32* ls_sorted_buffer_ptr[3]) {
	for (int counter_1 = 0; counter_1 < ITERATIONS_PER_THREAD; counter_1++) {
		const int OFFSET = blockDim.x * counter_1 + threadIdx.x;
		int ls_line[3];
		int result;
		unsigned char pivot_window[3][4];
		unsigned char* line = (unsigned char*)ls_line;

		ls_line[0] = ls_sorted_buffer_ptr[2][OFFSET];
		ls_line[1] = ls_sorted_buffer_ptr[2][OFFSET + 1];
		ls_line[2] = ls_sorted_buffer_ptr[2][OFFSET + 2];

		c_init_lower_pivot(line, pivot_window[0]);

		ls_line[0] = ls_sorted_buffer_ptr[1][OFFSET];
		ls_line[1] = ls_sorted_buffer_ptr[1][OFFSET + 1];
		ls_line[2] = ls_sorted_buffer_ptr[1][OFFSET + 2];

		c_init_median_pivot(line, pivot_window[1]);

		ls_line[0] = ls_sorted_buffer_ptr[0][OFFSET];
		ls_line[1] = ls_sorted_buffer_ptr[0][OFFSET + 1];
		ls_line[2] = ls_sorted_buffer_ptr[0][OFFSET + 2];

		c_init_high_pivot(line, pivot_window[2]);

		unsigned int sum = __vadd4(*((int*)(pivot_window[0])), __vadd4(*((int*)(pivot_window[1])), *((int*)(pivot_window[2]))));
		unsigned int min = __vminu4(*((int*)(pivot_window[0])), __vminu4(*((int*)(pivot_window[1])), *((int*)(pivot_window[2]))));
		unsigned int max = __vmaxu4(*((int*)(pivot_window[0])), __vmaxu4(*((int*)(pivot_window[1])), *((int*)(pivot_window[2]))));
		result = __vsub4(sum, __vadd4(min, max));

		ls_result_buffer[OFFSET] = result;
	}
}

inline __device__ 
void init_high_pivot(unsigned char* line,unsigned  char  pivot_window[4])
{
	for (int counter_2 = 3; counter_2 < 7; counter_2++) {
		unsigned int index = counter_2;
		if (line[counter_2] < line[counter_2 + 1]) {
			index = counter_2 + 1;
		}
		if (line[index] < line[counter_2 + 2]) {
			index = counter_2 + 2;
		}
		pivot_window[counter_2-3] = line[index];
	}
}

inline __device__ 
void init_median_pivot(unsigned char* line, unsigned  char pivot_window[4])
{
	for (int counter_2 = 3; counter_2 < 7; counter_2++) {
		unsigned int index_1 = counter_2;
		unsigned int index_2 = counter_2 + 1;
		if (line[counter_2] > line[counter_2 + 1]) {
			index_1 = counter_2 + 1;
			index_2 = counter_2;
		}
		unsigned int index_median = index_2;
		if (line[index_2] > line[counter_2 + 2]) {
			index_median = index_1;
			if (line[index_1] < line[counter_2+2]) {
				index_median = counter_2 + 2;
			}
			
		}
		pivot_window[counter_2-3] = line[index_median];
	}
}

inline __device__ 
void init_lower_pivot(unsigned char* line, unsigned char  pivot_window[4])
{
	for (int counter_2 = 3; counter_2 < 7; counter_2++) {
		unsigned int index = counter_2;
		if (line[counter_2] > line[counter_2 + 1]) {
			index = counter_2 + 1;
		}
		if (line[index] > line[counter_2 + 2]) {
			index = counter_2 + 2;
		}
		pivot_window[counter_2-3] = line[index];
	}
}

inline __device__
void c_init_high_pivot(unsigned char* line, unsigned  char  pivot_window[4])
{
	for (int counter_2 = 1; counter_2 < 5; counter_2++) {
		unsigned int index = counter_2;
		if (line[counter_2] < line[counter_2 + 3]) {
			index = counter_2 + 3;
		}
		if (line[index] < line[counter_2 + 6]) {
			index = counter_2 + 6;
		}
		pivot_window[counter_2 - 1] = line[index];
	}
}

inline __device__
void c_init_median_pivot(unsigned char* line, unsigned  char pivot_window[4])
{
	for (int counter_2 = 1; counter_2 < 5; counter_2++) {
		unsigned int index_1 = counter_2;
		unsigned int index_2 = counter_2 + 3;
		if (line[counter_2] > line[counter_2 + 3]) {
			index_1 = counter_2 + 3;
			index_2 = counter_2;
		}
		unsigned int index_median = index_2;
		if (line[index_2] > line[counter_2 + 6]) {
			index_median = index_1;
			if (line[index_1] < line[counter_2 + 6]) {
				index_median = counter_2 + 6;
			}

		}
		pivot_window[counter_2 - 1] = line[index_median];
	}
}

inline __device__
void c_init_lower_pivot(unsigned char* line, unsigned char  pivot_window[4])
{
	for (int counter_2 = 1; counter_2 < 5; counter_2++) {
		unsigned int index = counter_2;
		if (line[counter_2] > line[counter_2 + 3]) {
			index = counter_2 + 3;
		}
		if (line[index] > line[counter_2 + 6]) {
			index = counter_2 + 6;
		}
		pivot_window[counter_2 - 1] = line[index];
	}
}

inline __device__
void store(const unsigned __int32 counter, unsigned int* dst, unsigned int* ls_result_buffer, const unsigned __int32 BLOCK_OFFSET_X, const unsigned __int32 BLOCK_OFFSET_Y)
{
	const unsigned int BASE = SRC_PITCH_IN_INT * (BLOCK_OFFSET_Y + counter) + BLOCK_OFFSET_X;
	for (int counter_1 = 0; counter_1 < ITERATIONS_PER_THREAD; counter_1++) {
		const unsigned int OFFSET = counter_1 * blockDim.x + threadIdx.x;
		dst[BASE + OFFSET] = ls_result_buffer[OFFSET];
	}
	__syncthreads();
}

inline __device__ 
void sort(const unsigned int** ls_buffer_ptr, unsigned int** ls_sorted_buffer_ptr)
{
	unsigned int ls_column[3];
	unsigned int ls_sorted_column[3];
	char* column = (char*)ls_column;
	for (int counter = 0; counter < ITERATIONS_PER_THREAD; counter++) {
		const unsigned int INDEX = blockDim.x * counter + threadIdx.x;

		for (int counter_1 = 0; counter_1 < 3; counter_1++) {
			ls_column[counter_1] = ls_buffer_ptr[counter_1][INDEX];
		}

		ls_sorted_column[0] = __vminu4(ls_column[0], __vminu4(ls_column[1], ls_column[2]));
		ls_sorted_column[2] = __vmaxu4(ls_column[0], __vmaxu4(ls_column[1], ls_column[2]));
		unsigned int sum = __vadd4(ls_column[0], __vadd4(ls_column[1], ls_column[2]));
		ls_sorted_column[1] = __vsub4(__vsub4(sum, ls_sorted_column[2]), ls_sorted_column[0]);

		for (int counter_1 = 0; counter_1 < 3; counter_1++) {
			ls_sorted_buffer_ptr[counter_1][INDEX] = ls_sorted_column[counter_1];
		}
	}

	if (threadIdx.x < 2) {

		for (int counter_1 = 0; counter_1 < 3; counter_1++) {
			ls_column[counter_1] = ls_buffer_ptr[counter_1][BLOCK_LENGHT_IN_INT + threadIdx.x];
		}

		ls_sorted_column[0] = __vminu4(ls_column[0], __vminu4(ls_column[1], ls_column[2]));
		ls_sorted_column[2] = __vmaxu4(ls_column[0], __vmaxu4(ls_column[1], ls_column[2]));
		unsigned int sum = __vadd4(ls_column[0], __vadd4(ls_column[1], ls_column[2]));
		ls_sorted_column[1] = __vsub4(sum, __vadd4(ls_sorted_column[0], ls_sorted_column[2]));

		for (int counter_1 = 0; counter_1 < 3; counter_1++) {
			ls_sorted_buffer_ptr[counter_1][BLOCK_LENGHT_IN_INT + threadIdx.x] = ls_sorted_column[counter_1];
		}
	}
	__syncthreads();
}

inline __device__
void load_line(unsigned int* ls_buffer_ptr, const unsigned int* src, const signed int iter, const unsigned __int32 BLOCK_OFFSET_X, const unsigned __int32 BLOCK_OFFSET_Y)
{
	const unsigned __int32 y_offset = min((signed int)SRC_HEIGHT - 1, max(0, (signed int)(BLOCK_OFFSET_Y + iter)));
	const unsigned __int32 offset = SRC_PITCH_IN_INT * y_offset + BLOCK_OFFSET_X;
	for (int counter = 0; counter < ITERATIONS_PER_THREAD; counter++) {
		const unsigned int x_offset = threadIdx.x + blockDim.x * counter;
		ls_buffer_ptr[x_offset + 1] = src[offset + x_offset];
	}

	if (threadIdx.x == 0) {
		const int step = BLOCK_OFFSET_X == 0 ? 0 : -1;
		const int byte_y_offset = (SRC_PITCH_IN_INT * y_offset) << 2;
		unsigned int load_addr = 4 + BLOCK_LENGHT;
		unsigned int x_offset = (BLOCK_OFFSET_X + BLOCK_LENGHT_IN_INT) << 2;
		if (x_offset >= SRC_WIDTH) {
			load_addr = 4 + LAST_PIXEL_INDEX;
			x_offset = SRC_WIDTH - 1;
			
		}
		((unsigned char*)ls_buffer_ptr)[3] = ((unsigned char*)src)[byte_y_offset + (BLOCK_OFFSET_X << 2) + step];
		((unsigned char*)ls_buffer_ptr)[load_addr] = ((unsigned char*)src)[byte_y_offset + x_offset];
	}
	__syncthreads();
}

inline __device__
void c_load_line(unsigned int* ls_buffer_ptr, const unsigned int* src, const signed int iter, const unsigned __int32 BLOCK_OFFSET_X, const unsigned __int32 BLOCK_OFFSET_Y)
{
	const unsigned __int32 y_offset = min((signed int)SRC_HEIGHT - 1, max(0, (signed int)(BLOCK_OFFSET_Y + iter)));
	const unsigned __int32 offset = SRC_PITCH_IN_INT * y_offset + BLOCK_OFFSET_X;
	for (int counter = 0; counter < ITERATIONS_PER_THREAD; counter++) {
		const unsigned int x_offset = threadIdx.x + blockDim.x * counter;
		ls_buffer_ptr[x_offset + 1] = src[offset + x_offset];
	}

	if (threadIdx.x == 0) {
		const int byte_y_offset = (SRC_PITCH_IN_INT * y_offset) << 2;
		for (int counter = 0; counter < 3; counter++) {
			const int step = BLOCK_OFFSET_X == 0 ? counter : counter - 3;
			unsigned int load_addr = 4 + BLOCK_LENGHT + counter;
			unsigned int x_offset = ((BLOCK_OFFSET_X + BLOCK_LENGHT_IN_INT) << 2) + counter;
			if (x_offset >= SRC_WIDTH) {
				load_addr = 4 + LAST_PIXEL_INDEX + counter;
				x_offset = SRC_WIDTH - 3 + counter;
			}
			((unsigned char*)ls_buffer_ptr)[counter + 1] = ((unsigned char*)src)[byte_y_offset + (BLOCK_OFFSET_X << 2) + step];
			((unsigned char*)ls_buffer_ptr)[load_addr] = ((unsigned char*)src)[byte_y_offset + x_offset];
		}
	}
	__syncthreads();
}